#include "hip/hip_runtime.h"
#include "../../tools/config.cuh"
#include "../../tools/utilities.h"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <spdlog/spdlog.h>



// load opposite direction vectors for bounce-back, velocity direction vectors,
// and weight vectors into constant memory
// (fast, global, read-only lookup table identical for all threads)
// TODO: figure out how to safely use same constant memory across all .cu files
__constant__ int dvc_opp_dir[9];
__constant__ int dvc_c_x[9];
__constant__ int dvc_c_y[9];
__constant__ FP dvc_fp_c_x[9];
__constant__ FP dvc_fp_c_y[9];
__constant__ FP dvc_w[9];
bool constantsInitialized = false;
bool kernelAttributesDisplayed = false;

void InitializeConstants()
{
    if (constantsInitialized) { return; }

    //  0: ( 0,  0) = rest
    //  1: ( 1,  0) = east
    //  2: ( 0,  1) = north
    //  3: (-1,  0) = west
    //  4: ( 0, -1) = south
    //  5: ( 1,  1) = north-east
    //  6: (-1,  1) = north-west
    //  7: (-1, -1) = south-west
    //  8: ( 1, -1) = south-east

    // initialize opposite direction, velocity direction, and weight vectors
    int opp_dir[9] = { 0, 3, 4, 1, 2, 7, 8, 5, 6 };
    int c_x[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
    int c_y[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
    FP fp_c_x[9] = { 0.0,  1.0,  0.0, -1.0,  0.0,  1.0, -1.0, -1.0,  1.0 };
    FP fp_c_y[9] = { 0.0,  0.0,  1.0,  0.0, -1.0,  1.0,  1.0, -1.0, -1.0 };
    FP w[9] = { 4.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0,
                1.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0 };

    // copy them into constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_opp_dir), opp_dir, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_c_x), c_x, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_c_y), c_y, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_fp_c_x), fp_c_x, 9 * sizeof(FP));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_fp_c_y), fp_c_y, 9 * sizeof(FP));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_w), w, 9 * sizeof(FP));

    hipDeviceSynchronize();
    constantsInitialized = true;
}

// =============================================================================
// bounce-back boundary sub-kernels TODO: adjust for dealing with halo cells
// =============================================================================
__device__ __forceinline__ void ComputeNeighborIndex_BounceBackBoundary_Conditional_K(
    uint32_t src_x, uint32_t src_y, uint32_t src_y_global,
    uint32_t N_X, uint32_t N_Y_TOTAL,
    uint32_t i,
    uint32_t& dst_idx,
    uint32_t& dst_i)
{
    // check if directed into a wall
    if ((dvc_c_x[i] == -1 && src_x == 0) ||               // into left wall
        (dvc_c_x[i] ==  1 && src_x == N_X - 1) ||         // into right wall
        (dvc_c_y[i] == -1 && src_y == 0) ||               // into bottom wall
        (dvc_c_y[i] ==  1 && src_y == N_Y_TOTAL - 1))     // into top wall
    {
        // same cell but opposite direction because of bounce-back
        dst_idx = src_y * N_X + src_x;
        dst_i = dvc_opp_dir[i];
    }
    else
    {
        // normal neighbor in direction i
        dst_idx = (src_y + dvc_c_y[i]) * N_X + (src_x + dvc_c_x[i]);
        dst_i = i;
    }
}

__device__ __forceinline__ void ComputeNeighborIndex_BounceBackBoundary_BranchLess_K(
    uint32_t src_x, uint32_t src_y, uint32_t src_y_global,
    uint32_t N_X, uint32_t N_Y_TOTAL,
    uint32_t i,
    uint32_t& dst_idx,
    uint32_t& dst_i)
{
    // TODO: reduce register usage
    // branch-less bit-wise computation of: 1 if bounce-back, else 0
    int bounce =
        ((dvc_c_x[i] == -1) & (src_x == 0)) |
        ((dvc_c_x[i] ==  1) & (src_x == N_X - 1)) |
        ((dvc_c_y[i] == -1) & (src_y == 0)) |
        ((dvc_c_y[i] ==  1) & (src_y == N_Y_TOTAL - 1));

    // branch-less computation of destination index
    dst_idx = bounce * (src_y * N_X + src_x)
            + (1 - bounce) * ((src_y + dvc_c_y[i]) * N_X + (src_x + dvc_c_x[i]));

    // branch-less computation of destination direction
    dst_i = bounce * dvc_opp_dir[i] + (1 - bounce) * i;
}

// =============================================================================
// inject lid velocity sub-kernels TODO: adjust for dealing with halo cells
// =============================================================================
__device__ __forceinline__ void InjectLidVelocity_Conditional_K(
    uint32_t src_y,
    uint32_t N_Y,
    FP rho,
    FP omega,
    FP u_lid,
    uint32_t i,
    FP& f_new_i)
{
    // check if directed into the top wall
    if (dvc_c_y[i] == 1 && src_y == N_Y - 1)
    {
        f_new_i -= FP_CONST(6.0) * omega * rho * dvc_fp_c_x[i] * u_lid;
    }
}

__device__ __forceinline__ void InjectLidVelocity_BranchLess_K(
    uint32_t src_y_global,
    uint32_t N_Y_TOTAL,
    FP rho,
    FP omega,
    FP u_lid,
    uint32_t i,
    FP& f_new_i)
{
    // branch-less lid velocity injection via boolean mask
    int top_bounce = ((dvc_c_y[i] == 1) & (src_y_global == N_Y_TOTAL - 1));

    f_new_i -= top_bounce * FP_CONST(6.0) * dvc_w[i] * rho
             * dvc_fp_c_x[i] * u_lid;
}

// =============================================================================
// fully fused lattice update kernel for lid shear wave decay simulation
// =============================================================================
template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void FullyFusedLatticeUpdate_ShearWaveDecay_Push_K(
    const FP* const* __restrict__ dvc_df,
    FP* const* __restrict__ dvc_df_next,
    FP* const* __restrict__ dvc_df_halo_top,
    FP* const* __restrict__ dvc_df_halo_bottom,
    FP* __restrict__ dvc_rho,
    FP* __restrict__ dvc_u_x,
    FP* __restrict__ dvc_u_y,
    const FP omega,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_X_TOTAL, const uint32_t N_Y_TOTAL,
    const uint32_t Y_START, const uint32_t Y_END,
    const uint32_t N_CELLS,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // load df values into block-wise tiles of shared shared memory
    __shared__ FP tile_df[N_DIR][N_BLOCKSIZE];

    // used for summing stuff up and computing collision
    FP rho = FP_CONST(0.0);
    FP u_x = FP_CONST(0.0);
    FP u_y = FP_CONST(0.0);

    // populate shared memory tiles and compute sums in the same loop
    // density := sum over df values in each dir i
    // velocity := sum over df values, weighted by each dir i
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        tile_df[i][threadIdx.x] = dvc_df[i][idx];
        rho += tile_df[i][threadIdx.x];
        u_x += tile_df[i][threadIdx.x] * dvc_fp_c_x[i];
        u_y += tile_df[i][threadIdx.x] * dvc_fp_c_y[i];
    }

    // exit thread to avoid division by zero or erroneous values
    if (rho <= FP_CONST(0.0)) { return; }

    // finalize velocities
    u_x /= rho;
    u_y /= rho;

    // write back final field values only if requested
    if (write_rho) { dvc_rho[idx] = rho; }
    if (write_u_x) { dvc_u_x[idx] = u_x; }
    if (write_u_y) { dvc_u_y[idx] = u_y; }

    // pre-compute squared velocity and cell coordinates for this thread
    FP u_sq = u_x * u_x + u_y * u_y;
    uint32_t src_x = idx % N_X;
    uint32_t src_y = idx / N_X;
    uint32_t src_y_global = src_y + Y_START;

    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // compute dot product of c_i * u and equilibrium df value for dir i
        FP cu = dvc_fp_c_x[i] * u_x + dvc_fp_c_y[i] * u_y;
        FP f_eq_i = dvc_w[i] * rho
                  * (FP_CONST(1.0) + FP_CONST(3.0) * cu
                  + FP_CONST(4.5) * cu * cu - FP_CONST(1.5) * u_sq);

        // relax df towards equilibrium
        FP f_new_i = tile_df[i][threadIdx.x] - omega
                   * (tile_df[i][threadIdx.x] - f_eq_i);

        // TODO: inlined sub-kernel for the neighbor index
        // determine coordinates of the streaming destination cell
        // (with respect to periodic boundary conditions and halo cells)
        uint32_t dst_x = (src_x + dvc_c_x[i] + N_X) % N_X;
        uint32_t dst_y_raw = src_y + dvc_c_y[i]; // might not be within domain -> no %

        // check if streaming destination is outside of the process domain
        if (dst_y_raw < 0) // below -> stream into bottom halo
        {
            dvc_df_halo_bottom[i][dst_x] = f_new_i;
        }
        else if (dst_y_raw >= N_Y) // above -> stream into top halo
        {
            dvc_df_halo_top[i][dst_x] = f_new_i;
        }
        else // within -> stream to regular neighbor in regular df arrays
        {
            dvc_df_next[i][dst_y_raw * N_X + dst_x] = f_new_i;
        }
    }
}

// =============================================================================
// fully fused lattice update kernel for lid driven cavity simulation
// =============================================================================
template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void FullyFusedLatticeUpdate_LidDrivenCavity_Push_K(
    const FP* const* __restrict__ dvc_df,
    FP* const* __restrict__ dvc_df_next,
    FP* const* __restrict__ dvc_df_halo_top,
    FP* const* __restrict__ dvc_df_halo_bottom,
    FP* __restrict__ dvc_rho,
    FP* __restrict__ dvc_u_x,
    FP* __restrict__ dvc_u_y,
    const FP omega,
    const FP u_lid,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_X_TOTAL, const uint32_t N_Y_TOTAL,
    const uint32_t Y_START, const uint32_t Y_END,
    const uint32_t N_CELLS,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // load df values into block-wise tiles of shared shared memory
    // TODO: add 1-layer halo cells?
    __shared__ FP tile_df[N_DIR][N_BLOCKSIZE];

    // used for summing stuff up and computing collision
    FP rho = FP_CONST(0.0);
    FP u_x = FP_CONST(0.0);
    FP u_y = FP_CONST(0.0);

    // populate shared memory tiles and compute sums in the same loop
    // density := sum over df values in each dir i
    // velocity := sum over df values, weighted by each dir i
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        tile_df[i][threadIdx.x] = dvc_df[i][idx];
        rho += tile_df[i][threadIdx.x];
        u_x += tile_df[i][threadIdx.x] * dvc_fp_c_x[i];
        u_y += tile_df[i][threadIdx.x] * dvc_fp_c_y[i];
    }

    // exit thread to avoid division by zero or erroneous values
    if (rho <= FP_CONST(0.0)) { return; }

    // finalize velocities
    u_x /= rho;
    u_y /= rho;

    // write back final field values only if requested
    if (write_rho) { dvc_rho[idx] = rho; }
    if (write_u_x) { dvc_u_x[idx] = u_x; }
    if (write_u_y) { dvc_u_y[idx] = u_y; }

    // pre-compute squared velocity and cell coordinates for this thread
    FP u_sq = u_x * u_x + u_y * u_y;
    uint32_t src_x = idx % N_X;
    uint32_t src_y = idx / N_X;
    uint32_t src_y_global = src_y + Y_START;

    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // compute dot product of c_i * u and equilibrium df value for dir i
        FP cu = dvc_fp_c_x[i] * u_x + dvc_fp_c_y[i] * u_y;
        FP f_eq_i = dvc_w[i] * rho
                  * (FP_CONST(1.0) + FP_CONST(3.0) * cu
                  + FP_CONST(4.5) * cu * cu - FP_CONST(1.5) * u_sq);

        // relax df towards equilibrium
        FP f_new_i = tile_df[i][threadIdx.x] - omega
                   * (tile_df[i][threadIdx.x] - f_eq_i);

        // determine coordinates and direction of the streaming destination cell
        // (with respect to bounce-back boundary conditions and halo cells)
        // check if streaming is directed into a wall (bounce-back)
        if ((dvc_c_x[i] == -1 && src_x == 0) ||                  // into left wall
            (dvc_c_x[i] ==  1 && src_x == N_X - 1) ||            // into right wall
            (dvc_c_y[i] == -1 && src_y_global == 0) ||           // into bottom wall
            (dvc_c_y[i] ==  1 && src_y_global == N_Y_TOTAL - 1)) // into top wall
        {
            // inject lid velocity if streaming is directed into top wall
            if (dvc_c_y[i] == 1 && src_y_global == N_Y_TOTAL - 1)
            {
                f_new_i -= FP_CONST(6.0) * omega * rho * dvc_fp_c_x[i] * u_lid;
            }

            // same cell but opposite direction because of bounce-back
            // (definitely within the process domain -> stream into regular df arrays)
            dvc_df_next[dvc_opp_dir[i]][src_y * N_X + src_x] = f_new_i;
        }
        else // (might be outside of the process domain)
        {
            uint32_t dst_x_raw = src_x + dvc_c_x[i];
            uint32_t dst_y_raw = src_y + dvc_c_y[i];

            // check if streaming destination is outside of the process domain
            if (dst_y_raw < 0) // below, but no wall -> stream into bottom halo
            {
                dvc_df_halo_bottom[i][dst_x_raw] = f_new_i;
            }
            else if (dst_y_raw >= N_Y) // above, but no wall -> stream into top halo
            {
                dvc_df_halo_top[i][dst_x_raw] = f_new_i;
            }
            else // within -> stream to regular neighbor in regular df arrays
            {
                dvc_df_next[i][dst_y_raw * N_X + dst_x_raw] = f_new_i;
            }
        }
    }
}

void Launch_FullyFusedLatticeUpdate_Push(
    const FP* const* dvc_df,
    FP* const* dvc_df_next,
    FP* const* dvc_df_halo_top,
    FP* const* dvc_df_halo_bottom,
    FP* dvc_rho,
    FP* dvc_u_x,
    FP* dvc_u_y,
    const FP omega,
    const FP u_lid,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_X_TOTAL, const uint32_t N_Y_TOTAL,
    const uint32_t Y_START, const uint32_t Y_END,
    const uint32_t N_STEPS,
    const uint32_t N_CELLS,
    const uint32_t N_PROCESSES,
    const int RANK,
    const bool shear_wave_decay,
    const bool lid_driven_cavity,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    InitializeConstants();

    const uint32_t N_GRIDSIZE = (N_CELLS + N_BLOCKSIZE - 1) / N_BLOCKSIZE;

    // TODO: remove deprecated/unused kernel arguments
    if (shear_wave_decay)
    {
        FullyFusedLatticeUpdate_ShearWaveDecay_Push_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
            dvc_df, dvc_df_next, dvc_df_halo_top, dvc_df_halo_bottom,
            dvc_rho, dvc_u_x, dvc_u_y, omega, N_X, N_Y, N_X_TOTAL, N_Y_TOTAL,
            Y_START, Y_END, N_CELLS, write_rho, write_u_x, write_u_y);
    }
    else if (lid_driven_cavity)
    {
        FullyFusedLatticeUpdate_LidDrivenCavity_Push_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
            dvc_df, dvc_df_next, dvc_df_halo_top, dvc_df_halo_bottom,
            dvc_rho, dvc_u_x, dvc_u_y, omega, u_lid, N_X, N_Y, N_X_TOTAL, N_Y_TOTAL,
            Y_START, Y_END, N_CELLS, write_rho, write_u_x, write_u_y);
    }
    else
    {
        if (RANK == 0) { SPDLOG_ERROR("No valid simulation scenario selected"); }
    }

    // wait for GPU to finish operations
    hipDeviceSynchronize();

    if (RANK == 0 && !kernelAttributesDisplayed)
    {
        if (shear_wave_decay)
        {
            DisplayKernelAttributes(FullyFusedLatticeUpdate_ShearWaveDecay_Push_K<N_DIR, N_BLOCKSIZE>,
                fmt::format("FullyFusedLatticeUpdate_ShearWaveDecay_Push_K"),
                N_GRIDSIZE, N_BLOCKSIZE, N_X, N_Y, N_X_TOTAL, N_Y_TOTAL, N_STEPS, N_PROCESSES);
        }
        else if (lid_driven_cavity)
        {
            DisplayKernelAttributes(FullyFusedLatticeUpdate_LidDrivenCavity_Push_K<N_DIR, N_BLOCKSIZE>,
                fmt::format("FullyFusedLatticeUpdate_LidDrivenCavity_Push_K"),
                N_GRIDSIZE, N_BLOCKSIZE, N_X, N_Y, N_X_TOTAL, N_Y_TOTAL, N_STEPS, N_PROCESSES);
        }

        kernelAttributesDisplayed = true;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel of rank {} failed: {}",
            RANK, hipGetErrorString(err));
    }
}
