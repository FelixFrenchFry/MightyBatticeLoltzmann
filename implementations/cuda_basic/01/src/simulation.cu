#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <spdlog/spdlog.h>



// put velocity vectors into constant memory
// (fast, global, read-only lookup table identical for all threads)
__constant__ int dvc_vel_x[9];
__constant__ int dvc_vel_y[9];

// put the channel weights into constant memory
__constant__ float dvc_weights[9];

void Initialize()
{
    //  0: ( 0,  0) = rest
    //  1: ( 1,  0) = east
    //  2: ( 0,  1) = north
    //  3: (-1,  0) = west
    //  4: ( 0, -1) = south
    //  5: ( 1,  1) = north-east
    //  6: (-1,  1) = north-west
    //  7: (-1, -1) = south-west
    //  8: ( 1, -1) = south-east

    // initialize velocity vectors on the host
    int vel_x[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
    int vel_y[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };

    // initialize weights on the host
    float weights[9] = { 4.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f,
                         1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f };

    // copy them into constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_vel_x), vel_x, sizeof(vel_x));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_vel_y), vel_y, sizeof(vel_y));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_weights), weights, sizeof(weights));

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA Error: {}", hipGetErrorString(err));
    }
}

// ----- kernels -----

__global__ void InitializeDistributionFunction_K(float* dvc_distributionFunc,
                                                 float initValue,
                                                 int num_entries)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_entries) { return; }

    dvc_distributionFunc[index] = initValue;
}

__global__ void InitializeDensityField_K(float* densityField,
                                         float initValue,
                                         int num_entries)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_entries) { return; }

    densityField[index] = initValue;
}

__global__ void ComputeDensityField_K(const float* dvc_distributionFunc,
                                      float* dvc_densityField,
                                      int num_cells)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_cells) { return; }

    float densitySum = 0.0f;
    int base_index = index * 9;

    #pragma unroll
    for (int dir = 0; dir < 9; dir++)
    {
        densitySum += dvc_distributionFunc[base_index + dir];
    }

    dvc_densityField[index] = densitySum;
}

__global__ void ComputeVelocityField_K(const float* dvc_distributionFunc,
                                       const float* dvc_densityField,
                                       float* dvc_velocityField_x,
                                       float* dvc_velocityField_y,
                                       int num_cells)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_cells) { return; }

    float local_density = dvc_densityField[index];

    // exit thread if the cell has no mass and therefore no velocity
    if (local_density <= 0.0f)
    {
        dvc_velocityField_x[index] = 0.0f;
        dvc_velocityField_y[index] = 0.0f;
        return;
    }

    float velocitySum_x = 0.0f;
    float velocitySum_y = 0.0f;
    int base_index = index * 9;

    #pragma unroll
    for (int dir = 0; dir < 9; dir++)
    {
        velocitySum_x += dvc_distributionFunc[base_index + dir] * dvc_vel_x[dir];
        velocitySum_y += dvc_distributionFunc[base_index + dir] * dvc_vel_y[dir];
    }

    dvc_velocityField_x[index] = velocitySum_x / local_density;
    dvc_velocityField_y[index] = velocitySum_y / local_density;
}

__global__ void CollisionStep_K(float* dvc_distributionFunc,
                                const float* dvc_densityField,
                                const float* dvc_velocityField_x,
                                const float* dvc_velocityField_y,
                                float relaxOmega,
                                int num_cells)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_cells) { return; }

    int base_index = index * 9;

    // load into variables with proper notation for more readability
    float rho = dvc_densityField[index];
    float u_x = dvc_velocityField_x[index];
    float u_y = dvc_velocityField_y[index];
    float u_squ = u_x * u_x + u_y * u_y;

    // "collision" updates df values based on the density and velocity vals
    #pragma unroll
    for (int dir = 0; dir < 9; dir++)
    {
        // get direction vector and weight with proper notation
        int c_x = dvc_vel_x[dir];
        int c_y = dvc_vel_y[dir];
        float w = dvc_weights[dir];

        // dot product of c_i * u (directional velocity times local velocity)
        float cu = c_x * u_x + c_y * u_y;

        // compute equilibrium distribution f_eq
        float f_eq = w * rho * (1.0f + 3.0f * cu + 4.5f * cu * cu - 1.5f * u_squ);

        // relax distribution function towards equilibrium
        float f_i = dvc_distributionFunc[base_index + dir];
        dvc_distributionFunc[base_index + dir] = f_i + relaxOmega * (f_eq - f_i);
    }
}

// for each direction i, send distribution function component into that direction
// f_i(x, y) -> f_i(x + c_i.x, y + c_i.y)
__global__ void StreamingStep_K(const float* dvc_distributionFunc,
                                float* dvc_distributionFunc_next,
                                int grid_width, int grid_height,
                                int num_cells)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_cells) { return; }

    int cell_x = index % grid_width;
    int cell_y = index / grid_width;
    int base_index = index * 9;

    // "stream" (shift) distribution function components into each direction
    #pragma unroll
    for (int dir = 0; dir < 9; dir++)
    {
        // compute destiantion position, handling periodic boundary conditions
        int dest_x = (cell_x + dvc_vel_x[dir] + grid_width) % grid_width;
        int dest_y = (cell_y + dvc_vel_y[dir] + grid_height) % grid_height;
        int dest_index = (dest_y * grid_width + dest_x) * 9 + dir;

        // "stream" distributions to destination position
        dvc_distributionFunc_next[dest_index] = dvc_distributionFunc[base_index + dir];
    }
}

// ----- kernel launchers -----

void Launch_InitializeDistributionFunction_K(float* dvc_distributionFunc,
                                             float initValue,
                                             int num_entries)
{
    InitializeDistributionFunction_K<<<(num_entries + 255) / 256, 256>>>(
        dvc_distributionFunc, initValue, num_entries);

    // wait for device actions (kernels launch, memory copy, etc...) to finish
    // and report potential errors
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel error: {}", hipGetErrorString(err));
    }
}

void Launch_InitializeDensityField_K(float* densityField,
                                     float initValue,
                                     int num_entries)
{
    InitializeDensityField_K<<<(num_entries + 255) / 256, 256>>>(
        densityField, initValue, num_entries);

    // wait for device actions (kernels launch, memory copy, etc...) to finish
    // and report potential errors
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel error: {}", hipGetErrorString(err));
    }
}

void Launch_ComputeDensityField_K(const float* dvc_distributionFunc,
                                  float* dvc_densityField,
                                  int num_cells)
{
    ComputeDensityField_K<<<(num_cells + 255) / 256, 256>>>(
        dvc_distributionFunc, dvc_densityField, num_cells);

    // wait for device actions (kernels launch, memory copy, etc...) to finish
    // and report potential errors
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel error: {}", hipGetErrorString(err));
    }
}

void Launch_ComputeVelocityField_K(const float* dvc_distributionFunc,
                                   const float* dvc_densityField,
                                   float* dvc_velocityField_x,
                                   float* dvc_velocityField_y,
                                   int num_cells)
{
    ComputeVelocityField_K<<<(num_cells + 255) / 256, 256>>>(
        dvc_distributionFunc, dvc_densityField, dvc_velocityField_x,
        dvc_velocityField_y, num_cells);

    // wait for device actions (kernels launch, memory copy, etc...) to finish
    // and report potential errors
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel error: {}", hipGetErrorString(err));
    }
}

void Launch_CollisionStep_K(float* dvc_distributionFunc,
                            const float* dvc_densityField,
                            const float* dvc_velocityField_x,
                            const float* dvc_velocityField_y,
                            float relaxOmega,
                            int num_cells)
{
    CollisionStep_K<<<(num_cells + 255) / 256, 256>>>(
        dvc_distributionFunc, dvc_densityField, dvc_velocityField_x,
        dvc_velocityField_y, relaxOmega, num_cells);

    // wait for device actions (kernels launch, memory copy, etc...) to finish
    // and report potential errors
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel error: {}", hipGetErrorString(err));
    }
}

void Launch_StreamingStep_K(const float* dvc_distributionFunc,
                            float* dvc_distributionFunc_next,
                            int grid_width, int grid_height,
                            int num_cells)
{
    StreamingStep_K<<<(num_cells + 255) / 256, 256>>>(
        dvc_distributionFunc, dvc_distributionFunc_next, grid_width,
        grid_height, num_cells);

    // wait for device actions (kernels launch, memory copy, etc...) to finish
    // and report potential errors
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel error: {}", hipGetErrorString(err));
    }
}
