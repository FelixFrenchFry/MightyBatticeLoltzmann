#include "hip/hip_runtime.h"
#include "config.cuh"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <spdlog/spdlog.h>



// load velocity direction and weight vectors into constant memory
// (fast, global, read-only lookup table identical for all threads)
// TODO: figure out how to safely use same constant memory across all .cu files
__constant__ int dvc_ik_c_x[9];
__constant__ int dvc_ik_c_y[9];
__constant__ double dvc_ik_w[9];
bool constantsInitialized_IK = false;

void InitializeConstants_IK()
{
    if (constantsInitialized_IK) { return; }

    //  0: ( 0,  0) = rest
    //  1: ( 1,  0) = east
    //  2: ( 0,  1) = north
    //  3: (-1,  0) = west
    //  4: ( 0, -1) = south
    //  5: ( 1,  1) = north-east
    //  6: (-1,  1) = north-west
    //  7: (-1, -1) = south-west
    //  8: ( 1, -1) = south-east

    // initialize velocity direction and weight vectors on the host
    int c_x[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
    int c_y[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
    double w[9] = { 4.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0,
                    1.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0 };

    // copy them into constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_ik_c_x), c_x, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_ik_c_y), c_y, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_ik_w), w, 9 * sizeof(double));

    hipDeviceSynchronize();
    constantsInitialized_IK = true;
}

template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void ApplyShearWaveCondition_K(
    double* const* __restrict__ dvc_df,
    double* __restrict__ dvc_rho,
    double* __restrict__ dvc_u_x,
    double* __restrict__ dvc_u_y,
    const double rho_0,
    const double u_max,
    const double k,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // y-coordinate of the cell handled by this thread
    uint32_t y = idx / N_X;

    // compute sinusoidal x-velocity from the shear wave configuration
    double u_x_val = u_max * sin(k * static_cast<double>(y));
    double u_sq = u_x_val * u_x_val;

    // set initial values of the fields
    dvc_rho[idx] = rho_0;
    dvc_u_x[idx] = u_x_val;
    dvc_u_y[idx] = 0.0;

    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // compute dot product of c_i * u and equilibrium df value for dir i
        double cu = dvc_ik_c_x[i] * u_x_val + dvc_ik_c_y[i] * 0.0;
        double f_eq_i = dvc_ik_w[i] * rho_0
                      * (1.0 + 3.0 * cu + 4.5 * cu * cu - 1.5 * u_sq);

        // set initial df values
        dvc_df[i][idx] = f_eq_i;
    }
}

void Launch_ApplyShearWaveCondition_K(
    double* const* dvc_df,
    double* dvc_rho,
    double* dvc_u_x,
    double* dvc_u_y,
    const double rho_0,
    const double u_max,
    const double k,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS)
{
    InitializeConstants_IK();

    const uint32_t N_GRIDSIZE = (N_CELLS + N_BLOCKSIZE - 1) / N_BLOCKSIZE;

    ApplyShearWaveCondition_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
        dvc_df, dvc_rho, dvc_u_x, dvc_u_y, rho_0, u_max, k, N_X, N_Y, N_CELLS);

    // wait for GPU to finish operations
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel failed: {}", hipGetErrorString(err));
    }
}

template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void ApplyLidDrivenCavityCondition_K(
    double* const* __restrict__ dvc_df,
    double* __restrict__ dvc_rho,
    double* __restrict__ dvc_u_x,
    double* __restrict__ dvc_u_y,
    const double rho_0,
    const uint32_t N_CELLS)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // set initial values of the fields
    dvc_rho[idx] = rho_0;
    dvc_u_x[idx] = 0.0;
    dvc_u_y[idx] = 0.0;

    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // set initial df values
        dvc_df[i][idx] = dvc_ik_w[i] * rho_0;
    }
}

void Launch_ApplyLidDrivenCavityCondition_K(
    double* const* dvc_df,
    double* dvc_rho,
    double* dvc_u_x,
    double* dvc_u_y,
    const double rho_0,
    const uint32_t N_CELLS)
{
    InitializeConstants_IK();

    const uint32_t N_GRIDSIZE = (N_CELLS + N_BLOCKSIZE - 1) / N_BLOCKSIZE;

    ApplyLidDrivenCavityCondition_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
        dvc_df, dvc_rho, dvc_u_x, dvc_u_y, rho_0, N_CELLS);

    // wait for GPU to finish operations
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel failed: {}", hipGetErrorString(err));
    }
}
