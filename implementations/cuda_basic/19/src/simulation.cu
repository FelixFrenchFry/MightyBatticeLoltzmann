#include "hip/hip_runtime.h"
#include "../../tools/config.cuh"
#include "../../tools/utilities.h"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <spdlog/spdlog.h>



// load opposite direction vectors for bounce-back, velocity direction vectors,
// and weight vectors into constant memory
// (fast, global, read-only lookup table identical for all threads)
// TODO: figure out how to safely use same constant memory across all .cu files
__constant__ int dvc_opp_dir[9];
__constant__ int dvc_c_x[9];
__constant__ int dvc_c_y[9];
__constant__ FP dvc_fp_c_x[9];
__constant__ FP dvc_fp_c_y[9];
__constant__ FP dvc_w[9];
bool constantsInitialized = false;
bool kernelAttributesDisplayed = false;

void InitializeConstants()
{
    if (constantsInitialized) { return; }

    // ---------
    // | 6 2 5 |
    // | 3 0 1 |
    // | 7 4 8 |
    // ---------

    //  0: ( 0,  0) = rest
    //  1: ( 1,  0) = east
    //  2: ( 0,  1) = north
    //  3: (-1,  0) = west
    //  4: ( 0, -1) = south
    //  5: ( 1,  1) = north-east
    //  6: (-1,  1) = north-west
    //  7: (-1, -1) = south-west
    //  8: ( 1, -1) = south-east

    // initialize opposite direction, velocity direction, and weight vectors
    int opp_dir[9] = { 0, 3, 4, 1, 2, 7, 8, 5, 6 };
    int c_x[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
    int c_y[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
    FP fp_c_x[9] = { 0.0,  1.0,  0.0, -1.0,  0.0,  1.0, -1.0, -1.0,  1.0 };
    FP fp_c_y[9] = { 0.0,  0.0,  1.0,  0.0, -1.0,  1.0,  1.0, -1.0, -1.0 };
    FP w[9] = { 4.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0,
                1.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0 };

    // copy them into constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_opp_dir), opp_dir, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_c_x), c_x, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_c_y), c_y, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_fp_c_x), fp_c_x, 9 * sizeof(FP));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_fp_c_y), fp_c_y, 9 * sizeof(FP));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_w), w, 9 * sizeof(FP));

    hipDeviceSynchronize();
    constantsInitialized = true;
}

// =============================================================================
// fully fused lattice update kernel for shear wave decay sim
// =============================================================================
template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void FFLU_ShearWaveDecay_Push_K(
    const FP* const* __restrict__ dvc_df,
    FP* const* __restrict__ dvc_df_next,
    FP* __restrict__ dvc_rho,
    FP* __restrict__ dvc_u_x,
    FP* __restrict__ dvc_u_y,
    const FP omega,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // load df values into block-wise tiles of shared shared memory
    __shared__ FP tile_df[N_DIR][N_BLOCKSIZE];

    // used for summing stuff up and computing collision
    FP rho = FP_CONST(0.0);
    FP u_x = FP_CONST(0.0);
    FP u_y = FP_CONST(0.0);

    // populate shared memory tiles and compute sums in the same loop
    // density := sum over df values in each dir i
    // velocity := sum over df values, weighted by each dir i
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        tile_df[i][threadIdx.x] = dvc_df[i][idx];
        rho += tile_df[i][threadIdx.x];
        u_x += tile_df[i][threadIdx.x] * dvc_fp_c_x[i];
        u_y += tile_df[i][threadIdx.x] * dvc_fp_c_y[i];
    }

    // exit thread to avoid division by zero or erroneous values
    if (rho <= FP_CONST(0.0)) { return; }

    // finalize velocities
    u_x /= rho;
    u_y /= rho;

    // write back final field values only if requested
    if (write_rho) { dvc_rho[idx] = rho; }
    if (write_u_x) { dvc_u_x[idx] = u_x; }
    if (write_u_y) { dvc_u_y[idx] = u_y; }

    // pre-compute squared velocity and cell coordinates for this thread's cell
    FP u_sq = u_x * u_x + u_y * u_y;
    uint32_t src_x = idx % N_X;
    uint32_t src_y = idx / N_X;

    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // compute dot product of c_i * u and equilibrium df value for dir i
        FP cu = dvc_fp_c_x[i] * u_x + dvc_fp_c_y[i] * u_y;
        FP f_eq_i = dvc_w[i] * rho * (FP_CONST(1.0)
                  + FP_CONST(3.0) * cu
                  + FP_CONST(4.5) * cu * cu
                  - FP_CONST(1.5) * u_sq);

        // relax df towards equilibrium
        FP f_new_i = tile_df[i][threadIdx.x] - omega * (tile_df[i][threadIdx.x] - f_eq_i);

        // ---------
        // | 6 2 5 |
        // | 3 0 1 |
        // | 7 4 8 |
        // ---------
        // determine destination cell's index based on x/y-coordinates and dir i
        // (with respect to periodic boundary conditions)
        // TODO: sign bug in (uint32_t + int) math?
        uint32_t dst_idx = ((src_y + dvc_c_y[i] + N_Y) % N_Y) * N_X
                         + ((src_x + dvc_c_x[i] + N_X) % N_X);

        // stream df value to the destination in dir i
        dvc_df_next[i][dst_idx] = f_new_i;
    }
}

// =============================================================================
// fully fused lattice update kernel for shear wave decay sim (branchless)
// =============================================================================
template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void FFLU_ShearWaveDecay_Push_BL_K(
    const FP* const* __restrict__ dvc_df,
    FP* const* __restrict__ dvc_df_next,
    FP* __restrict__ dvc_rho,
    FP* __restrict__ dvc_u_x,
    FP* __restrict__ dvc_u_y,
    const FP omega,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // load df values into block-wise tiles of shared shared memory
    __shared__ FP tile_df[N_DIR][N_BLOCKSIZE];

    // used for summing stuff up and computing collision
    FP rho = FP_CONST(0.0);
    FP u_x = FP_CONST(0.0);
    FP u_y = FP_CONST(0.0);

    // populate shared memory tiles and compute sums in the same loop
    // density := sum over df values in each dir i
    // velocity := sum over df values, weighted by each dir i
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        tile_df[i][threadIdx.x] = dvc_df[i][idx];
        rho += tile_df[i][threadIdx.x];
        u_x += tile_df[i][threadIdx.x] * dvc_fp_c_x[i];
        u_y += tile_df[i][threadIdx.x] * dvc_fp_c_y[i];
    }

    // exit thread to avoid division by zero or erroneous values
    if (rho <= FP_CONST(0.0)) { return; }

    // finalize velocities
    u_x /= rho;
    u_y /= rho;

    // write back final field values only if requested
    if (write_rho) { dvc_rho[idx] = rho; }
    if (write_u_x) { dvc_u_x[idx] = u_x; }
    if (write_u_y) { dvc_u_y[idx] = u_y; }

    // pre-compute squared velocity and cell coordinates for this thread's cell
    FP u_sq = u_x * u_x + u_y * u_y;
    uint32_t src_x = idx % N_X;
    uint32_t src_y = idx / N_X;

    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // compute dot product of c_i * u and equilibrium df value for dir i
        FP cu = dvc_fp_c_x[i] * u_x + dvc_fp_c_y[i] * u_y;
        FP f_eq_i = dvc_w[i] * rho * (FP_CONST(1.0)
                  + FP_CONST(3.0) * cu
                  + FP_CONST(4.5) * cu * cu
                  - FP_CONST(1.5) * u_sq);

        // relax df towards equilibrium
        FP f_new_i = tile_df[i][threadIdx.x] - omega * (tile_df[i][threadIdx.x] - f_eq_i);

        // determine destination cell's index based on x/y-coordinates and dir i
        // (with respect to periodic boundary conditions)
        // ---------
        // | 6 2 5 |
        // | 3 0 1 |
        // | 7 4 8 |
        // ---------
        // TODO: sign bug in (uint32_t + int) math?
        uint32_t dst_idx = ((src_y + dvc_c_y[i] + N_Y) % N_Y) * N_X
                         + ((src_x + dvc_c_x[i] + N_X) % N_X);

        // stream df value to the destination in dir i
        dvc_df_next[i][dst_idx] = f_new_i;
    }
}

// =============================================================================
// fully fused lattice update kernel for lid driven cavity sim
// =============================================================================
template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void FFLU_LidDrivenCavity_Push_K(
    const FP* const* __restrict__ dvc_df,
    FP* const* __restrict__ dvc_df_next,
    FP* __restrict__ dvc_rho,
    FP* __restrict__ dvc_u_x,
    FP* __restrict__ dvc_u_y,
    const FP omega,
    const FP u_lid,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // load df values into block-wise tiles of shared shared memory
    __shared__ FP tile_df[N_DIR][N_BLOCKSIZE];

    // used for summing stuff up and computing collision
    FP rho = FP_CONST(0.0);
    FP u_x = FP_CONST(0.0);
    FP u_y = FP_CONST(0.0);

    // populate shared memory tiles and compute sums in the same loop
    // density := sum over df values in each dir i
    // velocity := sum over df values, weighted by each dir i
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        tile_df[i][threadIdx.x] = dvc_df[i][idx];
        rho += tile_df[i][threadIdx.x];
        u_x += tile_df[i][threadIdx.x] * dvc_fp_c_x[i];
        u_y += tile_df[i][threadIdx.x] * dvc_fp_c_y[i];
    }

    // exit thread to avoid division by zero or erroneous values
    if (rho <= FP_CONST(0.0)) { return; }

    // finalize velocities
    u_x /= rho;
    u_y /= rho;

    // write back final field values only if requested
    if (write_rho) { dvc_rho[idx] = rho; }
    if (write_u_x) { dvc_u_x[idx] = u_x; }
    if (write_u_y) { dvc_u_y[idx] = u_y; }

    // pre-compute squared velocity and cell coordinates for this thread's cell
    FP u_sq = u_x * u_x + u_y * u_y;
    uint32_t src_x = idx % N_X;
    uint32_t src_y = idx / N_X;

    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // compute dot product of c_i * u and equilibrium df value for dir i
        FP cu = dvc_fp_c_x[i] * u_x + dvc_fp_c_y[i] * u_y;
        FP f_eq_i = dvc_w[i] * rho * (FP_CONST(1.0)
                  + FP_CONST(3.0) * cu
                  + FP_CONST(4.5) * cu * cu
                  - FP_CONST(1.5) * u_sq);

        // relax df towards equilibrium
        FP f_new_i = tile_df[i][threadIdx.x] - omega * (tile_df[i][threadIdx.x] - f_eq_i);

        // determine coordinates and direction of the streaming destination cell
        // (with respect to bounce-back boundary conditions)
        // ---------
        // | 6 2 5 |
        // | 3 0 1 |
        // | 7 4 8 |
        // ---------
        // check if streaming is directed into a wall (bounce-back)
        if ((dvc_c_x[i] == -1 && src_x == 0) ||        // into left wall
            (dvc_c_x[i] ==  1 && src_x == N_X - 1) ||  // into right wall
            (dvc_c_y[i] == -1 && src_y == 0) ||        // into bottom wall
            (dvc_c_y[i] ==  1 && src_y == N_Y - 1))    // into top wall
        {
            // inject lid velocity if streaming is directed into top wall
            if (dvc_c_y[i] == 1 && src_y == N_Y - 1)
            {
                // TODO: correct equation w.r.t. omega and dvc_w[i] ?
                f_new_i -= FP_CONST(6.0) * dvc_w[i] * rho * dvc_fp_c_x[i] * u_lid;
            }

            // same cell but opposite direction of dir i because of bounce-back
            dvc_df_next[dvc_opp_dir[i]][src_y * N_X + src_x] = f_new_i;
        }
        else // not directed into a wall
        {
            // stream df value to the destination in regular dir i
            dvc_df_next[i][(src_y + dvc_c_y[i]) * N_X + (src_x + dvc_c_x[i])] = f_new_i;
        }
    }
}

// =============================================================================
// fully fused lattice update kernel for lid driven cavity sim (branchless)
// =============================================================================
template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void FFLU_LidDrivenCavity_Push_BL_K(
    const FP* const* __restrict__ dvc_df,
    FP* const* __restrict__ dvc_df_next,
    FP* __restrict__ dvc_rho,
    FP* __restrict__ dvc_u_x,
    FP* __restrict__ dvc_u_y,
    const FP omega,
    const FP u_lid,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // load df values into block-wise tiles of shared shared memory
    __shared__ FP tile_df[N_DIR][N_BLOCKSIZE];

    // used for summing stuff up and computing collision
    FP rho = FP_CONST(0.0);
    FP u_x = FP_CONST(0.0);
    FP u_y = FP_CONST(0.0);

    // populate shared memory tiles and compute sums in the same loop
    // density := sum over df values in each dir i
    // velocity := sum over df values, weighted by each dir i
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        tile_df[i][threadIdx.x] = dvc_df[i][idx];
        rho += tile_df[i][threadIdx.x];
        u_x += tile_df[i][threadIdx.x] * dvc_fp_c_x[i];
        u_y += tile_df[i][threadIdx.x] * dvc_fp_c_y[i];
    }

    // exit thread to avoid division by zero or erroneous values
    if (rho <= FP_CONST(0.0)) { return; }

    // finalize velocities
    u_x /= rho;
    u_y /= rho;

    // write back final field values only if requested
    if (write_rho) { dvc_rho[idx] = rho; }
    if (write_u_x) { dvc_u_x[idx] = u_x; }
    if (write_u_y) { dvc_u_y[idx] = u_y; }

    // pre-compute squared velocity and cell coordinates for this thread's cell
    FP u_sq = u_x * u_x + u_y * u_y;
    uint32_t src_x = idx % N_X;
    uint32_t src_y = idx / N_X;

    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // compute dot product of c_i * u and equilibrium df value for dir i
        FP cu = dvc_fp_c_x[i] * u_x + dvc_fp_c_y[i] * u_y;
        FP f_eq_i = dvc_w[i] * rho * (FP_CONST(1.0)
                  + FP_CONST(3.0) * cu
                  + FP_CONST(4.5) * cu * cu
                  - FP_CONST(1.5) * u_sq);

        // relax df towards equilibrium
        FP f_new_i = tile_df[i][threadIdx.x] - omega * (tile_df[i][threadIdx.x] - f_eq_i);

        // determine coordinates and direction of the streaming destination cell
        // (with respect to bounce-back boundary conditions)
        // ---------
        // | 6 2 5 |
        // | 3 0 1 |
        // | 7 4 8 |
        // ---------
        // branchless bit-wise computation of wall collision:
        // 1 if bounce-back, else 0
        uint32_t bounce =
            ((dvc_c_x[i] == -1) & (src_x == 0)) |        // into left wall
            ((dvc_c_x[i] ==  1) & (src_x == N_X - 1)) |  // into right wall
            ((dvc_c_y[i] == -1) & (src_y == 0)) |        // into bottom wall
            ((dvc_c_y[i] ==  1) & (src_y == N_Y - 1));   // into top wall

        // branchless computation of destination index
        uint32_t dst_idx = bounce * (src_y * N_X + src_x)
                         + (1 - bounce) * ((src_y + dvc_c_y[i]) * N_X + (src_x + dvc_c_x[i]));

        // branchless computation of destination direction
        uint32_t dst_i = bounce * dvc_opp_dir[i] + (1 - bounce) * i;

        // branchless lid velocity injection via boolean mask
        uint32_t top_bounce = ((dvc_c_y[i] == 1) & (src_y == N_Y - 1));
        f_new_i -= top_bounce
                 * FP_CONST(6.0) * dvc_w[i] * rho * dvc_fp_c_x[i] * u_lid;

        // stream df value to the destination in dir i
        // (dir i got reversed in case of bounce-back)
        dvc_df_next[dst_i][dst_idx] = f_new_i;
    }
}

void Launch_FullyFusedLatticeUpdate_Push(
    const FP* const* dvc_df,
    FP* const* dvc_df_next,
    FP* dvc_rho,
    FP* dvc_u_x,
    FP* dvc_u_y,
    const FP omega,
    const FP u_lid,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_STEPS,
    const uint32_t N_CELLS,
    const bool shear_wave_decay,
    const bool lid_driven_cavity,
    const bool branchless,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    InitializeConstants();

    const uint32_t N_GRIDSIZE = (N_CELLS + N_BLOCKSIZE - 1) / N_BLOCKSIZE;

    if (shear_wave_decay && !branchless)
    {
        FFLU_ShearWaveDecay_Push_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
            dvc_df, dvc_df_next, dvc_rho, dvc_u_x, dvc_u_y, omega,
            N_X, N_Y, N_CELLS, write_rho, write_u_x, write_u_y);
    }
    else if (shear_wave_decay && branchless)
    {
        FFLU_ShearWaveDecay_Push_BL_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
            dvc_df, dvc_df_next, dvc_rho, dvc_u_x, dvc_u_y, omega,
            N_X, N_Y, N_CELLS, write_rho, write_u_x, write_u_y);
    }
    else if (lid_driven_cavity && !branchless)
    {
        FFLU_LidDrivenCavity_Push_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
            dvc_df, dvc_df_next, dvc_rho, dvc_u_x, dvc_u_y, omega, u_lid,
            N_X, N_Y, N_CELLS, write_rho, write_u_x, write_u_y);
    }
    else if (lid_driven_cavity && branchless)
    {
        FFLU_LidDrivenCavity_Push_BL_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
            dvc_df, dvc_df_next, dvc_rho, dvc_u_x, dvc_u_y, omega, u_lid,
            N_X, N_Y, N_CELLS, write_rho, write_u_x, write_u_y);
    }
    else
    {
        SPDLOG_ERROR("No valid simulation scenario selected");
    }

    // wait for GPU to finish operations
    hipDeviceSynchronize();

    if (!kernelAttributesDisplayed)
    {
        if (shear_wave_decay && !branchless)
        {
            DisplayKernelAttributes(FFLU_ShearWaveDecay_Push_K<N_DIR, N_BLOCKSIZE>,
                fmt::format("FFLU_ShearWaveDecay_Push_K"),
                N_GRIDSIZE, N_BLOCKSIZE, N_X, N_Y);
        }
        else if (shear_wave_decay && branchless)
        {
            DisplayKernelAttributes(FFLU_ShearWaveDecay_Push_BL_K<N_DIR, N_BLOCKSIZE>,
                fmt::format("FFLU_ShearWaveDecay_Push_BL_K"),
                N_GRIDSIZE, N_BLOCKSIZE, N_X, N_Y);
        }
        else if (lid_driven_cavity && !branchless)
        {
            DisplayKernelAttributes(FFLU_LidDrivenCavity_Push_K<N_DIR, N_BLOCKSIZE>,
                fmt::format("FFLU_LidDrivenCavity_Push_K"),
                N_GRIDSIZE, N_BLOCKSIZE, N_X, N_Y);
        }
        else if (lid_driven_cavity && branchless)
        {
            DisplayKernelAttributes(FFLU_LidDrivenCavity_Push_BL_K<N_DIR, N_BLOCKSIZE>,
                fmt::format("FFLU_LidDrivenCavity_Push_BL_K"),
                N_GRIDSIZE, N_BLOCKSIZE, N_X, N_Y);
        }

        kernelAttributesDisplayed = true;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        // specify detailed logging for the error message
        spdlog::set_pattern("[%Y-%m-%d %H:%M:%S] [%s:%#] [%^%l%$] %v");

        SPDLOG_ERROR("CUDA error: {}\n", hipGetErrorString(err));

        // return to basic logging
        spdlog::set_pattern("[%Y-%m-%d %H:%M:%S] [%^%l%$] %v");
    }
}
