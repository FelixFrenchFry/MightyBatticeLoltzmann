#include "hip/hip_runtime.h"
#include "../../tools/config.cuh"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <spdlog/spdlog.h>



// load opposite direction vectors for bounce-back, velocity direction vectors,
// and weight vectors into constant memory
// (fast, global, read-only lookup table identical for all threads)
// TODO: figure out how to safely use same constant memory across all .cu files
__constant__ int dvc_opp_dir[9];
__constant__ int dvc_c_x[9];
__constant__ int dvc_c_y[9];
__constant__ float dvc_w[9];
bool constantsInitialized = false;

void InitializeConstants()
{
    if (constantsInitialized) { return; }

    //  0: ( 0,  0) = rest
    //  1: ( 1,  0) = east
    //  2: ( 0,  1) = north
    //  3: (-1,  0) = west
    //  4: ( 0, -1) = south
    //  5: ( 1,  1) = north-east
    //  6: (-1,  1) = north-west
    //  7: (-1, -1) = south-west
    //  8: ( 1, -1) = south-east

    // initialize opposite direction, velocity direction, and weight vectors
    int opp_dir[9] = { 0, 3, 4, 1, 2, 7, 8, 5, 6 };
    int c_x[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
    int c_y[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
    float w[9] = { 4.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f,
                   1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f };

    // copy them into constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_opp_dir), opp_dir, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_c_x), c_x, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_c_y), c_y, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_w), w, 9 * sizeof(float));

    hipDeviceSynchronize();
    constantsInitialized = true;
}

template <uint32_t N_DIR>
__device__ __forceinline__ uint32_t ComputeDensityAndVelocity_K(
    const float* const* __restrict__ dvc_df,
    uint32_t idx,
    float& rho, float& u_x, float& u_y)
{
    rho = 0.0f;
    u_x = 0.0f;
    u_y = 0.0f;

    // density := sum over df values in each dir i
    // velocity := sum over df values, weighted by each dir i
    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        float df_i = dvc_df[i][idx];
        rho += df_i;
        u_x += df_i * dvc_c_x[i];
        u_y += df_i * dvc_c_y[i];
    }
}

__device__ __forceinline__ void ComputeNeighborIndex_PeriodicBoundary_K(
    uint32_t src_x, uint32_t src_y,
    uint32_t N_X, uint32_t N_Y,
    uint32_t i,
    uint32_t& dst_idx,
    uint32_t& dst_i)
{
    // determine index of the destination neihbor cell
    // (with respect to periodic boundary conditions)
    dst_idx = ((src_y + dvc_c_y[i] + N_Y) % N_Y) * N_X
            + ((src_x + dvc_c_x[i] + N_X) % N_X);
    dst_i = i;
}

__device__ __forceinline__ void ComputeNeighborIndex_BounceBackBoundary_Conditional_K(
    uint32_t src_x, uint32_t src_y,
    uint32_t N_X, uint32_t N_Y,
    uint32_t i,
    uint32_t& dst_idx,
    uint32_t& dst_i)
{
    // check if directed into a wall
    if ((dvc_c_x[i] == -1 && src_x == 0) ||        // into left wall
        (dvc_c_x[i] ==  1 && src_x == N_X - 1) ||  // into right wall
        (dvc_c_y[i] == -1 && src_y == 0) ||        // into bottom wall
        (dvc_c_y[i] ==  1 && src_y == N_Y - 1))    // into top wall
    {
        // same cell but opposite direction because of bounce-back
        dst_idx = src_y * N_X + src_x;
        dst_i = dvc_opp_dir[i];
    }
    else
    {
        // normal neighbor in direction i
        dst_idx = (src_y + dvc_c_y[i]) * N_X + (src_x + dvc_c_x[i]);
        dst_i = i;
    }
}

__device__ __forceinline__ void ComputeNeighborIndex_BounceBackBoundary_BranchLess_K(
    uint32_t src_x, uint32_t src_y,
    uint32_t N_X, uint32_t N_Y,
    uint32_t i,
    uint32_t& dst_idx,
    uint32_t& dst_i)
{
    // TODO: this increases register pressure by too much

    // branch-less bit-wise computation of: 1 if bounce-back, else 0
    int bounce =
        ((dvc_c_x[i] == -1) & (src_x == 0)) |
        ((dvc_c_x[i] ==  1) & (src_x == N_X - 1)) |
        ((dvc_c_y[i] == -1) & (src_y == 0)) |
        ((dvc_c_y[i] ==  1) & (src_y == N_Y - 1));

    // branch-less computation of destination index
    // TODO: reduce register usage
    /*
    uint32_t idx_normal = (src_y + dvc_c_y[i]) * N_X + (src_x + dvc_c_x[i]);
    uint32_t idx_bounce = src_y * N_X + src_x;
    dst_idx = bounce * idx_bounce + (1 - bounce) * idx_normal;
    */
    dst_idx = bounce * (src_y * N_X + src_x)
            + (1 - bounce) * ((src_y + dvc_c_y[i]) * N_X + (src_x + dvc_c_x[i]));

    // branch-less computation of destination direction
    dst_i = bounce * dvc_opp_dir[i] + (1 - bounce) * i;
}

__device__ __forceinline__ void InjectLidVelocity_Conditional_K(
    uint32_t src_y,
    uint32_t N_Y,
    float rho,
    float omega,
    float u_lid,
    uint32_t i,
    float& f_new_i)
{
    // check if directed into top wall
    if (dvc_c_y[i] == 1 && src_y == N_Y - 1)
    {
        f_new_i -= 6.0f * omega * rho * dvc_c_x[i] * u_lid;
    }
}

__device__ __forceinline__ void InjectLidVelocity_BranchLess_K(
    uint32_t src_y,
    uint32_t N_Y,
    float rho,
    float omega,
    float u_lid,
    uint32_t i,
    float& f_new_i)
{
    // branch-less lid velocity injection via boolean mask
    int top_bounce = ((dvc_c_y[i] == 1) & (src_y == N_Y - 1));

    f_new_i -= top_bounce * 6.0f * dvc_w[i] * rho
             * static_cast<float>(dvc_c_x[i]) * u_lid;
}

template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void ComputeFullyFusedOperations_K(
    const float* const* __restrict__ dvc_df,
    float* const* __restrict__ dvc_df_next,
    float* __restrict__ dvc_rho,
    float* __restrict__ dvc_u_x,
    float* __restrict__ dvc_u_y,
    const float omega,
    const float u_lid,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // inlined sub-kernel for the density and velocity
    float rho, u_x, u_y;
    ComputeDensityAndVelocity_K<N_DIR>(
        dvc_df, idx, rho, u_x, u_y);

    // exit thread to avoid division by zero or erroneous values
    if (rho <= 0.0f) { return; }

    // finalize velocities
    u_x /= rho;
    u_y /= rho;

    // write back final field values only if requested
    if (write_rho) { dvc_rho[idx] = rho; }
    if (write_u_x) { dvc_u_x[idx] = u_x; }
    if (write_u_y) { dvc_u_y[idx] = u_y; }

    // pre-compute squared velocity and cell coordinates for this thread
    float u_sq = u_x * u_x + u_y * u_y;
    uint32_t src_x = idx % N_X;
    uint32_t src_y = idx / N_X;

    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // compute dot product of c_i * u and equilibrium df value for dir i
        float cu = static_cast<float>(dvc_c_x[i]) * u_x
                 + static_cast<float>(dvc_c_y[i]) * u_y;
        float f_eq_i = dvc_w[i] * rho
                     * (1.0f + 3.0f * cu + 4.5f * cu * cu - 1.5f * u_sq);

        // relax df towards equilibrium
        float f_new_i = dvc_df[i][idx] - omega * (dvc_df[i][idx] - f_eq_i);

        // inlined sub-kernel for the neighbor index
        uint32_t dst_idx, dst_i;
        ComputeNeighborIndex_PeriodicBoundary_K(
            src_x, src_y, N_X, N_Y, i, dst_idx, dst_i);

        // inject lid velocity if directed into top wall
        //InjectLidVelocity_BranchLess_K(src_y, N_Y, rho, omega, u_lid, i,
        //    f_new_i);

        // stream df value df_i to the neighbor in dir i
        // (direction i gets reversed in case of bounce-back)
        dvc_df_next[dst_i][dst_idx] = f_new_i;
    }
}

void Launch_FullyFusedOperationsComputation(
    const float* const* dvc_df,
    float* const* dvc_df_next,
    float* dvc_rho,
    float* dvc_u_x,
    float* dvc_u_y,
    const float omega,
    const float u_lid,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_STEPS,
    const uint32_t N_CELLS,
    const bool write_rho,
    const bool write_u_x,
    const bool write_u_y)
{
    InitializeConstants();

    const uint32_t N_GRIDSIZE = (N_CELLS + N_BLOCKSIZE - 1) / N_BLOCKSIZE;

    ComputeFullyFusedOperations_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
        dvc_df, dvc_df_next, dvc_rho, dvc_u_x, dvc_u_y, omega, u_lid, N_X, N_Y,
        N_CELLS, write_rho, write_u_x, write_u_y);

    // wait for GPU to finish operations
    hipDeviceSynchronize();

    if (!kernelAttributesDisplayed)
    {
        DisplayKernelAttributes(ComputeFullyFusedOperations_K<N_DIR, N_BLOCKSIZE>,
            fmt::format("ComputeFullyFusedOperations_K"),
            N_GRIDSIZE, N_BLOCKSIZE, N_X, N_Y, N_STEPS);

        kernelAttributesDisplayed = true;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA kernel failed: {}", hipGetErrorString(err));
    }
}
