#include "hip/hip_runtime.h"
#include "config.cuh"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <spdlog/spdlog.h>
#include <stdio.h>



// load velocity direction and weight vectors into constant memory
// (fast, global, read-only lookup table identical for all threads)
// TODO: figure out how to safely use same constant memory across all .cu files
__constant__ int dvc_ck_c_x[9];
__constant__ int dvc_ck_c_y[9];
__constant__ float dvc_ck_w[9];
bool constantsInitialized_CK = false;

void InitializeConstants_CK()
{
    // one-time initialization guard
    if (constantsInitialized_CK) { return; }

    //  0: ( 0,  0) = rest
    //  1: ( 1,  0) = east
    //  2: ( 0,  1) = north
    //  3: (-1,  0) = west
    //  4: ( 0, -1) = south
    //  5: ( 1,  1) = north-east
    //  6: (-1,  1) = north-west
    //  7: (-1, -1) = south-west
    //  8: ( 1, -1) = south-east

    // initialize velocity direction and weight vectors on the host
    int c_x[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
    int c_y[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
    float w[9] = { 4.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f,
                   1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f };

    // copy them into constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_ck_c_x), c_x, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_ck_c_y), c_y, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_ck_w), w, 9 * sizeof(float));

    hipDeviceSynchronize();
    constantsInitialized_CK = true;
}

template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void ComputeCollision_K(
    float* const* __restrict__ dvc_df,
    const float* __restrict__ dvc_rho,
    const float* __restrict__ dvc_u_x,
    const float* __restrict__ dvc_u_y,
    const float omega,
    const uint32_t N_CELLS)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // declare and populate df_i in shared memory tile like df_tile[i][thread]
    __shared__ float df_tile[N_DIR][N_BLOCKSIZE];
    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        df_tile[i][threadIdx.x] = dvc_df[i][idx];
    }
    // wait for data to be loaded
    __syncthreads();

    // load temp variables into read-only cache and multiple loads
    float rho = __ldg(&dvc_rho[idx]);
    float u_x = __ldg(&dvc_u_x[idx]);
    float u_y = __ldg(&dvc_u_y[idx]);
    float u_sq = u_x * u_x + u_y * u_y;

    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // temp variables for better readability
        float c_x = static_cast<float>(dvc_ck_c_x[i]);
        float c_y = static_cast<float>(dvc_ck_c_y[i]);
        float w = dvc_ck_w[i];

        // dot product of c_i * u (velocity directions times local velocity)
        float cu = c_x * u_x + c_y * u_y;
        float cu2 = cu * cu;

        // compute equilibrium distribution f_eq_i for current direction i
        float f_eq_i = w * rho * (1.0f + 3.0f * cu + 4.5f * cu2 - 1.5f * u_sq);

        // relax distribution function towards equilibrium
        float f_i = df_tile[i][threadIdx.x];
        dvc_df[i][idx] = f_i + omega * (f_eq_i - f_i);
    }
}

void Launch_CollisionComputation(
    float* const* dvc_df,
    const float* dvc_rho,
    const float* dvc_u_x,
    const float* dvc_u_y,
    const float omega,
    const uint32_t N_CELLS)
{
    InitializeConstants_CK();

    const uint32_t N_GRIDSIZE = (N_CELLS + N_BLOCKSIZE - 1) / N_BLOCKSIZE;

    ComputeCollision_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
        dvc_df, dvc_rho, dvc_u_x, dvc_u_y, omega, N_CELLS);

    // wait for device actions to finish and report potential errors
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("Kernel '{}' failed at line {}: {}",
                     __func__, __LINE__, hipGetErrorString(err));
    }
}
