#include "hip/hip_runtime.h"
#include "config.cuh"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <spdlog/spdlog.h>
#include <stdio.h>



// load velocity direction vectors into constant memory
// (fast, global, read-only lookup table identical for all threads)
// TODO: figure out how to safely use same constant memory across all .cu files
__constant__ int dvc_vk_c_x[9];
__constant__ int dvc_vk_c_y[9];
bool constantsInitialized_VK = false;

void InitializeConstants_VK()
{
    // one-time initialization guard
    if (constantsInitialized_VK) { return; }

    //  0: ( 0,  0) = rest
    //  1: ( 1,  0) = east
    //  2: ( 0,  1) = north
    //  3: (-1,  0) = west
    //  4: ( 0, -1) = south
    //  5: ( 1,  1) = north-east
    //  6: (-1,  1) = north-west
    //  7: (-1, -1) = south-west
    //  8: ( 1, -1) = south-east

    // initialize velocity direction vectors on the host
    int c_x[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
    int c_y[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };

    // copy them into constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_vk_c_x), c_x, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_vk_c_y), c_y, 9 * sizeof(int));

    hipDeviceSynchronize();
    constantsInitialized_VK = true;
}

// __restriced__ tells compiler there is no overlap among the data pointed to
// (reduces memory access and instructions, but increases register pressure!)
template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void ComputeVelocityField_K(
    const float* const* __restrict__ dvc_df,
    const float* __restrict__ dvc_rho,
    float* __restrict__ dvc_u_x,
    float* __restrict__ dvc_u_y,
    const uint32_t N_CELLS)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // declare and populate df_i in shared memory tile like df_tile[i][thread]
    __shared__ float df_tile[N_DIR][N_BLOCKSIZE];
    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        df_tile[i][threadIdx.x] = dvc_df[i][idx];
    }
    // wait for data to be loaded
    __syncthreads();

    // load temp variable into read-only cache and multiple loads
    float rho = __ldg(&dvc_rho[idx]);

    // exit thread to avoid division by zero or erroneous values
    if (rho <= 0.0f)
    {
        dvc_u_x[idx] = 0.0f;
        dvc_u_y[idx] = 0.0f;
        return;
    }

    float sum_x = 0.0f;
    float sum_y = 0.0f;

    // sum over distribution function values, weighted by each direction i
    // (SoA layout for coalesced memory access across threads)
    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // load data from shared memory tile with local index
        float df_i = df_tile[i][threadIdx.x];
        sum_x += df_i * dvc_vk_c_x[i];
        sum_y += df_i * dvc_vk_c_y[i];
    }

    // divide sums by density to obtain final velocities
    dvc_u_x[idx] = sum_x / rho;
    dvc_u_y[idx] = sum_y / rho;
}

void Launch_VelocityFieldComputation(
    const float* const* dvc_df,
    const float* dvc_rho,
    float* dvc_u_x,
    float* dvc_u_y,
    const uint32_t N_CELLS)
{
    InitializeConstants_VK();

    const uint32_t N_GRIDSIZE = (N_CELLS + N_BLOCKSIZE - 1) / N_BLOCKSIZE;

    ComputeVelocityField_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
        dvc_df, dvc_rho, dvc_u_x, dvc_u_y, N_CELLS);

    // wait for device actions to finish and report potential errors
    hipDeviceSynchronize();

    // debugging helper
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("CUDA velocity kernel error: {}",
            hipGetErrorString(err));
    }
}
