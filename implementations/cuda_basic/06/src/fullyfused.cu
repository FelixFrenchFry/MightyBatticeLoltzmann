#include "hip/hip_runtime.h"
#include "../../tools/config.cuh"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <spdlog/spdlog.h>



// load velocity direction and weight vectors into constant memory
// (fast, global, read-only lookup table identical for all threads)
__constant__ int dvc_c_x[9];
__constant__ int dvc_c_y[9];
__constant__ float dvc_w[9];
bool constantsInitialized = false;

void InitializeConstants()
{
    // one-time initialization guard
    if (constantsInitialized) { return; }

    //  0: ( 0,  0) = rest
    //  1: ( 1,  0) = east
    //  2: ( 0,  1) = north
    //  3: (-1,  0) = west
    //  4: ( 0, -1) = south
    //  5: ( 1,  1) = north-east
    //  6: (-1,  1) = north-west
    //  7: (-1, -1) = south-west
    //  8: ( 1, -1) = south-east

    // initialize velocity direction and weight vectors on the host
    int c_x[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
    int c_y[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
    float w[9] = { 4.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f, 1.0f/9.0f,
                   1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f, 1.0f/36.0f };

    // copy them into constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_c_x), c_x, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_c_y), c_y, 9 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dvc_w), w, 9 * sizeof(float));

    hipDeviceSynchronize();
    constantsInitialized = true;
}

__device__ __forceinline__ float& get_df_component(DF_Vec& vec, int i)
{
    float* p = reinterpret_cast<float*>(&vec);
    return p[i - 1];
}

// __restriced__ tells compiler there is no overlap among the data pointed to
// (reduces memory access and instructions, but increases register pressure!)
template <uint32_t N_DIR, uint32_t N_BLOCKSIZE>
__global__ void ComputeFullyFusedOperations_K(
    const DF_Vec* __restrict__ dvc_df_1_to_8,
    DF_Vec* __restrict__ dvc_df_next_1_to_8,
    float* __restrict__ dvc_df_0,
    float* __restrict__ dvc_rho,
    float* __restrict__ dvc_u_x,
    float* __restrict__ dvc_u_y,
    const float omega,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_CELLS) { return; }

    // declare and populate df_i in shared memory tile like df_tile[i][thread]
    __shared__ float df_tile[N_DIR][N_BLOCKSIZE];

    // vectorized load of first batch of 4 values into shared memory
    df_tile[1][threadIdx.x] = dvc_df_1_to_8[idx].df_1_to_4.x;
    df_tile[2][threadIdx.x] = dvc_df_1_to_8[idx].df_1_to_4.y;
    df_tile[3][threadIdx.x] = dvc_df_1_to_8[idx].df_1_to_4.z;
    df_tile[4][threadIdx.x] = dvc_df_1_to_8[idx].df_1_to_4.w;

    // vectorized load of second batch of 4 values into shared memory
    df_tile[5][threadIdx.x] = dvc_df_1_to_8[idx].df_5_to_8.x;
    df_tile[6][threadIdx.x] = dvc_df_1_to_8[idx].df_5_to_8.y;
    df_tile[7][threadIdx.x] = dvc_df_1_to_8[idx].df_5_to_8.z;
    df_tile[8][threadIdx.x] = dvc_df_1_to_8[idx].df_5_to_8.w;

    // separate non-vectorized laod of the center value
    df_tile[0][threadIdx.x] = dvc_df_0[idx];

    // wait for data to be fully loaded
    __syncthreads();

    // ----- DENSITY COMPUTATION -----

    // used initially as sum and later as final velocities in computations
    float rho = 0.0f;

    // sum over distribution function values in each direction i
    // (SoA layout for coalesced memory access across threads)
    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // load data from shared memory tile with local index
        rho += df_tile[i][threadIdx.x];
    }

    dvc_rho[idx] = rho;

    // ----- VELOCITY COMPUTATION -----

    // exit thread to avoid division by zero or erroneous values
    if (rho <= 0.0f)
    {
        dvc_u_x[idx] = 0.0f;
        dvc_u_y[idx] = 0.0f;
        return;
    }

    // used initially as sums and later as final velocities in computations
    float u_x = 0.0f;
    float u_y = 0.0f;

    // sum over distribution function values, weighted by each direction i
    // (SoA layout for coalesced memory access across threads)
    #pragma unroll
    for (uint32_t i = 0; i < N_DIR; i++)
    {
        // load data from shared memory tile with local index
        float df_i = df_tile[i][threadIdx.x];
        u_x += df_i * dvc_c_x[i];
        u_y += df_i * dvc_c_y[i];
    }

    // divide sums by density to obtain final velocities
    u_x /= rho;
    u_y /= rho;
    dvc_u_x[idx] = u_x;
    dvc_u_y[idx] = u_y;

    // ----- COLLISION AND STREAMING COMPUTATION -----

    // load temp variables into read-only cache and multiple loads
    float u_sq = u_x * u_x + u_y * u_y;

    // determine coordinates of the source cell handled by this thread
    // TODO: bug in coordinate computation?
    uint32_t src_x = idx % N_X;
    uint32_t src_y = idx / N_X;

    #pragma unroll // TODO: limit unroll for lower register pressure
    for (uint32_t i = 1; i < N_DIR; i++)
    {
        // dot product of c_i * u (velocity directions times local velocity)
        float cu = static_cast<float>(dvc_c_x[i]) * u_x
                 + static_cast<float>(dvc_c_y[i]) * u_y;

        // compute equilibrium distribution f_eq_i for current direction i
        float f_eq_i = dvc_w[i] * rho
                     * (1.0f + 3.0f * cu + 4.5f * cu * cu - 1.5f * u_sq);

        // relax distribution function towards equilibrium
        // TODO: bug in this optimized computation?
        float f_new_i = df_tile[i][threadIdx.x] * (1 - omega) + omega * f_eq_i;

        // determine coordinates and index within the SoA of the target cell
        // (with respect to periodic boundary conditions)
        uint32_t dst_idx = ((src_y + dvc_c_y[i] + N_Y) % N_Y) * N_X
                         + ((src_x + dvc_c_x[i] + N_X) % N_X);

        // stream distribution function value df_i to neighbor in direction i
        // TODO: bug bug bug
        get_df_component(dvc_df_next_1_to_8[dst_idx], i) = f_new_i;
    }

    // separate update of the center value in different data structure
    float f_eq_0 = (4.0f/9.0f) * rho * (1.0f - 1.5f * u_sq);
    float f_eq_new_0 = df_tile[0][threadIdx.x] * (1 - omega) + omega * f_eq_0;
    dvc_df_0[idx] = f_eq_new_0;
}

void Launch_FullyFusedOperationsComputation(
    const DF_Vec* dvc_df_1_to_8,
    DF_Vec* dvc_df_next_1_to_8,
    float* dvc_df_0,
    float* dvc_rho,
    float* dvc_u_x,
    float* dvc_u_y,
    const float omega,
    const uint32_t N_X, const uint32_t N_Y,
    const uint32_t N_CELLS)
{
    InitializeConstants();

    const uint32_t N_GRIDSIZE = (N_CELLS + N_BLOCKSIZE - 1) / N_BLOCKSIZE;

    ComputeFullyFusedOperations_K<N_DIR, N_BLOCKSIZE><<<N_GRIDSIZE, N_BLOCKSIZE>>>(
        dvc_df_1_to_8, dvc_df_next_1_to_8, dvc_df_0, dvc_rho, dvc_u_x, dvc_u_y,
        omega, N_X, N_Y, N_CELLS);

    // wait for device actions to finish and report potential errors
    hipDeviceSynchronize();

    if (!kernelAttributesDisplayed)
    {
        DisplayKernelAttributes(ComputeFullyFusedOperations_K<N_DIR, N_BLOCKSIZE>,
            fmt::format("ComputeFullyFusedOperations_K<{}, {}>", N_DIR, N_BLOCKSIZE));

        kernelAttributesDisplayed = true;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        SPDLOG_ERROR("Kernel '{}' failed: {}",
                     __func__, hipGetErrorString(err));
    }
}
